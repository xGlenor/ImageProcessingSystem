#include "hip/hip_runtime.h"
#include "image_kernels.h"
#include <cstdio>
#include <cstring>

__global__ void grayscaleKernel(unsigned char* img, int w, int h, int c) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= w || y >= h) return;
    int idx = (y * w + x) * c;
    unsigned char r = img[idx], g = img[idx+1], b = img[idx+2];
    unsigned char gray = (unsigned char)((r+g+b)/3);
    img[idx] = img[idx+1] = img[idx+2] = gray;
}

__global__ void blurKernel(
    const unsigned char* d_in,
    unsigned char* d_out,
    int w, int h, int c)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (x >= w || y >= h) return;

    int idx = (y*w + x)*c;
    int sum[3] = {0,0,0};
    int count = 0;

    for(int dy=-1; dy<=1; ++dy){
        for(int dx=-1; dx<=1; ++dx){
            int nx = x + dx, ny = y + dy;
            if (nx>=0 && nx<w && ny>=0 && ny<h){
                int nidx = (ny*w + nx)*c;
                sum[0] += d_in[nidx + 0];
                sum[1] += d_in[nidx + 1];
                sum[2] += d_in[nidx + 2];
                ++count;
            }
        }
    }
    // uśredniamy
    d_out[idx + 0] = sum[0] / count;
    d_out[idx + 1] = sum[1] / count;
    d_out[idx + 2] = sum[2] / count;
}

extern "C"
__declspec(dllexport)
bool ProcessImage(
    unsigned char* imageData, int width, int height, int channels,
    const char* filterName, double* gpuTimeMs
) {
    size_t numBytes = width * height * channels * sizeof(unsigned char);
    unsigned char* d_img = nullptr;

    // Alokacja GPU
    if (hipMalloc(&d_img, numBytes) != hipSuccess) return false;
    if (hipMemcpy(d_img, imageData, numBytes, hipMemcpyHostToDevice) != hipSuccess) {
        hipFree(d_img); return false;
    }

    // Ustawienie siatki
    dim3 block(16, 16);
    dim3 grid((width+15)/16, (height+15)/16);

    // Pomiary czasu GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Wybór kernela
    if (strcmp(filterName, "grayscale") == 0) {
        grayscaleKernel<<<grid, block>>>(d_img, width, height, channels);
    }
    else if (strcmp(filterName, "blur") == 0) {
        blurKernel<<<grid, block>>>(d_img, d_img, width, height, channels);
    }
    else {
        // domyślnie grayscale
        grayscaleKernel<<<grid, block>>>(d_img, width, height, channels);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    *gpuTimeMs = ms;

    // Kopiowanie z powrotem
    hipMemcpy(imageData, d_img, numBytes, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_img);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return true;
}
